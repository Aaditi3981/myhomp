#include "hip/hip_runtime.h"
/*
 * Rectangular matrix multiplication, started from MIT Cilk matmul.cilk example
 *
 */
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "omp.h"
#include <pthread.h>
#include <string.h>
#define REAL float
#include "libxomp.h" 
#include "xomp_cuda_lib_inlined.cu"
#include "homp.h"

void zero(float *A,int n)
{
  int i;
  int j;
{
    int _p_i;
    int _p_j;
    long p_index_;
    long p_lower_;
    long p_upper_;
    XOMP_loop_default(0,n - 1,1,&p_lower_,&p_upper_);
    for (p_index_ = p_lower_; p_index_ <= p_upper_; p_index_ += 1) {
      for (_p_j = 0; _p_j < n; _p_j++) {
        A[(p_index_ * n) + _p_j] = 0.0;
      }
    }
    XOMP_barrier();
  }
}

void init(float *A,int n)
{
  int i;
  int j;
{
    int _p_i;
    int _p_j;
    long p_index_;
    long p_lower_;
    long p_upper_;
    XOMP_loop_default(0,n - 1,1,&p_lower_,&p_upper_);
    for (p_index_ = p_lower_; p_index_ <= p_upper_; p_index_ += 1) {
      for (_p_j = 0; _p_j < n; _p_j++) {
        A[(p_index_ * n) + _p_j] = (drand48());
      }
    }
    XOMP_barrier();
  }
}

double maxerror(float *A,float *B,int n)
{
  int i;
  int j;
  double error = 0.0;
  for (i = 0; i < n; i++) {
    for (j = 0; j < n; j++) {
      double diff = ((A[(i * n) + j] - B[(i * n) + j]) / A[(i * n) + j]);
//        printf("%4f -- %4f\n", A[i*n+j], B[i*n+j]);
      if (diff < 0) 
        diff = -diff;
      if (diff > error) 
        error = diff;
    }
  }
  return error;
}

void iter_matmul(float *A,float *B,float *C,int n)
{
  int i;
  int j;
  int k;
  for (i = 0; i < n; i++) 
    for (k = 0; k < n; k++) {
      float c = 0.0;
      for (j = 0; j < n; j++) 
        c += (A[(i * n) + j] * B[(j * n) + k]);
      C[(i * n) + k] = c;
    }
}

struct OUT__3__7117___data 
{
  void *A_p;
  void *B_p;
  void *C_p;
  void *n_p;
}
;
static void OUT__3__7117__(void *__out_argv);

void omp_matmul(float *A,float *B,float *C,int n)
{
  int i;
  int j;
  int k;
  struct OUT__3__7117___data __out_argv2__7117__;
  __out_argv2__7117__.n_p = ((void *)(&n));
  __out_argv2__7117__.C_p = ((void *)(&C));
  __out_argv2__7117__.B_p = ((void *)(&B));
  __out_argv2__7117__.A_p = ((void *)(&A));
  XOMP_parallel_start(OUT__3__7117__,&__out_argv2__7117__,1,0,"/data/yy8/2013-8-multiple-gpu-work/benchmarks/matrixmultiplication/matmul.c",73);
  XOMP_parallel_end("/data/yy8/2013-8-multiple-gpu-work/benchmarks/matrixmultiplication/matmul.c",80);
}

void openacc_matmul(float *A,float *B,float *C,int n)
{
  int i;
  int j;
  int k;
/* #pragma acc kernels copyin(A[0:n][0:n],B[0:n][0:n]) copyout(C[0:n][0:n]) */
//#pragma acc kernels loop copyin(A[0:n*n],B[0:n*n]) copyout(C[0:n*n])
  
#pragma acc parallel loop copyin ( A [ 0 : n * n ], B [ 0 : n * n ] ) copyout ( C [ 0 : n * n ] ) collapse ( 2 )
  for (i = 0; i < n; i++) 
    for (k = 0; k < n; k++) {
      float c = 0.0;
      for (j = 0; j < n; j++) 
        c += (A[(i * n) + j] * B[(j * n) + k]);
      C[(i * n) + k] = c;
    }
}

struct OUT__1__7117___data 
{
  void *n_p;
  void *num_threads_p;
  void *A_p;
  void *B_p;
  void *C_seq_p;
  void *C_omp_for_p;
  void *C_acc_p;
}
;
static void OUT__1__7117__(void *__out_argv);
void matmul_ompacc_mdev_v1(REAL *A, REAL *B, REAL *C,  int n);
void matmul_ompacc_mdev_v2(REAL *A, REAL *B, REAL *C,  int n);
void matmul_ompacc_mdev_v3(REAL *A, REAL *B, REAL *C,  int n);

int main(int argc,char *argv[])
{
  int n;
  int num_threads;
  float *A;
  float *B;
  float *C_seq;
  float *C_omp_for;
  float *C_acc;
  double seq_elapsed;
  double omp_for_elapsed;
  double acc_elapsed;
  if (argc != 2) {
    fprintf(stderr,"Usage: matmul <n> [<1|2|3>]\n");
    fprintf(stderr,"\t 1: row dist; 2: column dist; 3: both row/column dist; default 1\n");
    exit(1);
  }
  n = atoi(argv[1]);
  int dist = 1;
  if (argc == 3) dist = atoi(argv[2]);
  A = ((float *)(malloc(((n * n) * sizeof(float )))));
  B = ((float *)(malloc(((n * n) * sizeof(float )))));
  C_seq = ((float *)(malloc(((n * n) * sizeof(float )))));
  C_omp_for = ((float *)(malloc(((n * n) * sizeof(float )))));
  C_acc = ((float *)(malloc(((n * n) * sizeof(float )))));
  srand48((1 << 12));
  struct OUT__1__7117___data __out_argv1__7117__;
  __out_argv1__7117__.C_acc_p = ((void *)(&C_acc));
  __out_argv1__7117__.C_omp_for_p = ((void *)(&C_omp_for));
  __out_argv1__7117__.C_seq_p = ((void *)(&C_seq));
  __out_argv1__7117__.B_p = ((void *)(&B));
  __out_argv1__7117__.A_p = ((void *)(&A));
  __out_argv1__7117__.num_threads_p = ((void *)(&num_threads));
  __out_argv1__7117__.n_p = ((void *)(&n));
  XOMP_parallel_start(OUT__1__7117__,&__out_argv1__7117__,1,0,"/data/yy8/2013-8-multiple-gpu-work/benchmarks/matrixmultiplication/matmul.c",152);
  XOMP_parallel_end("/data/yy8/2013-8-multiple-gpu-work/benchmarks/matrixmultiplication/matmul.c",163);
/* sequential run */
  seq_elapsed = omp_get_wtime();
  iter_matmul(A, B, C_seq, n);
  seq_elapsed = (omp_get_wtime() - seq_elapsed);
/* openmp parallel for version */
  omp_for_elapsed = omp_get_wtime();
  omp_matmul(A, B, C_omp_for, n);
  omp_for_elapsed = (omp_get_wtime() - omp_for_elapsed);
/* we currently cannot do the OpenMP acc and OpenACC run in once */
#ifndef OPENACC
/* openmp acc version */
  omp_init_devices();
  acc_elapsed = omp_get_wtime();
  if (dist == 2)
	  matmul_ompacc_mdev_v2(A,B,C_acc,n);
  else if (dist == 3)
	  matmul_ompacc_mdev_v3(A,B,C_acc,n);
  else
	  matmul_ompacc_mdev_v1(A,B,C_acc,n);
  acc_elapsed = (omp_get_wtime() - acc_elapsed);
#else
#endif
  printf("=======================================================================\n");
  printf("\t\tmatmul(%dx%d) example on %d threads(cores)\n",n,n,num_threads);
  printf("-----------------------------------------------------------------------\n");
  printf("Performance:  Runtime (s)\t MFLOPS\t\t\t Error\n");
  printf("-----------------------------------------------------------------------\n");
  printf("Sequential      :  %4f \t\t %4f\t\t%g\n",seq_elapsed,((((2.0 * n) * n) * n) / (1.0e6 * seq_elapsed)),maxerror(C_seq,C_seq,n));
  printf("OMP For         :  %4f \t\t %4f\t\t%g\n",omp_for_elapsed,((((2.0 * n) * n) * n) / (1.0e6 * omp_for_elapsed)),maxerror(C_seq,C_omp_for,n));
#ifndef OPENACC
  printf("OMP ACC         :  %4f \t\t %4f\t\t%g\n",acc_elapsed,((((2.0 * n) * n) * n) / (1.0e6 * acc_elapsed)),maxerror(C_seq,C_acc,n));
#else
#endif
  free(C_acc);
  free(C_omp_for);
  free(C_seq);
  free(B);
  free(A);
  return 0;
}

static void OUT__1__7117__(void *__out_argv)
{
  int *n = (int *)(((struct OUT__1__7117___data *)__out_argv) -> n_p);
  int *num_threads = (int *)(((struct OUT__1__7117___data *)__out_argv) -> num_threads_p);
  float **A = (float **)(((struct OUT__1__7117___data *)__out_argv) -> A_p);
  float **B = (float **)(((struct OUT__1__7117___data *)__out_argv) -> B_p);
  float **C_seq = (float **)(((struct OUT__1__7117___data *)__out_argv) -> C_seq_p);
  float **C_omp_for = (float **)(((struct OUT__1__7117___data *)__out_argv) -> C_omp_for_p);
  float **C_acc = (float **)(((struct OUT__1__7117___data *)__out_argv) -> C_acc_p);
  if (XOMP_master()) {
     *num_threads = omp_get_num_threads();
  }
  init( *A, *n);
  init( *B, *n);
  zero( *C_seq, *n);
  zero( *C_omp_for, *n);
  zero( *C_acc, *n);
}

static void OUT__3__7117__(void *__out_argv)
{
  float **A = (float **)(((struct OUT__3__7117___data *)__out_argv) -> A_p);
  float **B = (float **)(((struct OUT__3__7117___data *)__out_argv) -> B_p);
  float **C = (float **)(((struct OUT__3__7117___data *)__out_argv) -> C_p);
  int *n = (int *)(((struct OUT__3__7117___data *)__out_argv) -> n_p);
  int _p_i;
  int _p_j;
  int _p_k;
  long p_index_;
  long p_lower_;
  long p_upper_;
  XOMP_loop_default(0, *n - 1,1,&p_lower_,&p_upper_);
  for (p_index_ = p_lower_; p_index_ <= p_upper_; p_index_ += 1) {
    for (_p_k = 0; _p_k <  *n; _p_k++) {
      float c = 0.0;
      for (_p_j = 0; _p_j <  *n; _p_j++) 
        c += (( *A)[(p_index_ *  *n) + _p_j] * ( *B)[(_p_j *  *n) + _p_k]);
      ( *C)[(p_index_ *  *n) + _p_k] = c;
    }
  }
  XOMP_barrier();
}

#if 0
/**
 * CUDA threading is through the k-dimension
 * */
__global__ void OUT__2__7117_mdev_v2__(int start_k, int length_k, int n,float *_dev_A,float *_dev_B,float *_dev_C)
{
  int _p_i;
  int _p_j;
  int _p_k;
  int _dev_k = blockDim.x * blockIdx.x + threadIdx.x;
  if (_dev_k >= start_k && _dev_k <= length_k - 1) {
	for (_p_i=0; _p_i<n; _p_i++) {
      float c = 0.0;
      for (_p_j = 0; _p_j < n; _p_j++)
        c += (_dev_A[(_p_i * n) + _p_j] * _dev_B[(_p_j * length_k) + _dev_k]);
      _dev_C[(_p_i * length_k) + _dev_k] = c;
    }
  }
}
#endif

/**
 * The unified mdev kernel, which CUDA threading is always from the dim0 of A
 * CUDA threading is through the i-dimension
 * A[N_i][N_j]
 * B[N_j][N_k]
 * C[N_i][N_k]
 * A*B=C
 */
__global__ void OUT__2__7117_mdev__(int N_i, int N_j, int N_k, float *_dev_A,float *_dev_B,float *_dev_C)
{
  int _p_i;
  int _p_j;
  int _p_k;
  int _dev_i = blockDim.x * blockIdx.x + threadIdx.x;
  if (_dev_i >= 0 && _dev_i <= N_i - 1) {
    for (_p_k = 0; _p_k < N_k; _p_k++) {
      float c = 0.0;
      for (_p_j = 0; _p_j < N_j; _p_j++)
        c += (_dev_A[(_dev_i * N_j) + _p_j] * _dev_B[(_p_j * N_k) + _p_k]);
      _dev_C[(_dev_i * N_k) + _p_k] = c;
    }
  }
}


#if 0
/* multiple device */

/* A, C row-major partition */
void ompacc_matmul_mdev_v1(REAL *A, REAL *B, REAL *C, int n)
{
    int i, j, k;
#pragma omp target device(*) map(from:C[0:n]{0:n}>>(*)), map(to:n,A[0:n]{0:n}>>(*),B[0:n][0:n])
#pragma omp parallel for private(i,j,k) dist_iteration match_range C[:]
    for (i = 0; i < n; i++)
        for (k = 0; k < n; k++) {
            REAL c = 0.0;
            for (j = 0; j < n; j++)
                c += A[i * n + j] * B[j * n + k];
            C[i * n + k] = c;
        }
}
#endif

void matmul_ompacc_mdev_v1(REAL *A, REAL *B, REAL *C,  int n)
{
	   /* get number of target devices specified by the programmers */
	    int __num_target_devices__ = omp_get_num_active_devices(); /*XXX: = runtime or compiler generated code */

	    printf("use %d target devices\n", __num_target_devices__);

		omp_device_t *__target_devices__[__num_target_devices__];
		/**TODO: compiler generated code or runtime call to init the __target_devices__ array */
		int __i__;
		for (__i__ = 0; __i__ < __num_target_devices__; __i__++) {
			__target_devices__[__i__] = &omp_devices[__i__]; /* currently this is simple a copy of the pointer */
		}
		/**TODO: compiler generated code or runtime call to init the topology */
		int __top_ndims__ = 1;
		int __top_dims__[__top_ndims__];
		omp_factor(__num_target_devices__, __top_dims__, __top_ndims__);
		int __top_periodic__[__top_ndims__]; __top_periodic__[0] = 0;
		omp_grid_topology_t __topology__={__num_target_devices__, __top_ndims__, __top_dims__, __top_periodic__};
		omp_grid_topology_t *__topp__ = &__topology__;

		int __num_mapped_variables__ = 3; /* XXX: need compiler output */

		omp_stream_t __dev_stream__[__num_target_devices__]; /* need to change later one for omp_stream_t struct */
		omp_data_map_info_t __data_map_infos__[__num_mapped_variables__];

		omp_data_map_info_t * __info__ = &__data_map_infos__[0];
		omp_data_map_init_info(__info__, __topp__, A, sizeof(REAL), OMP_MAP_TO, n, n, 1);
		__info__->maps = (omp_data_map_t **)alloca(sizeof(omp_data_map_t *) * __num_target_devices__);

		__info__ = &__data_map_infos__[1];
		omp_data_map_init_info(__info__, __topp__, B, sizeof(REAL), OMP_MAP_TO, n, n, 1);
		__info__->maps = (omp_data_map_t **)alloca(sizeof(omp_data_map_t *) * __num_target_devices__);

		__info__ = &__data_map_infos__[2];
		omp_data_map_init_info(__info__, __topp__, C, sizeof(REAL), OMP_MAP_FROM, n, n, 1);
		__info__->maps = (omp_data_map_t **)alloca(sizeof(omp_data_map_t *) * __num_target_devices__);

		omp_data_map_t __data_maps__[__num_target_devices__][__num_mapped_variables__];
		for (__i__ = 0; __i__ < __num_target_devices__; __i__++) {
	    		printf("=========================================== device %d ==========================================\n", __i__);
			omp_device_t * __dev__ = __target_devices__[__i__];
			omp_set_current_device(__dev__);
			omp_init_stream(__dev__, &__dev_stream__[__i__]);

			/***************** for each mapped variable has to and tofrom, if it has region mapped to this __ndev_i__ id, we need code here *******************************/
			omp_data_map_t * __dev_map_A__ = &__data_maps__[__i__][0]; /* 0 is given by compiler here */
			omp_data_map_init_map(__dev_map_A__, &__data_map_infos__[0], __i__, __dev__, &__dev_stream__[__i__]);
			omp_data_map_do_even_map(__dev_map_A__, 0, __topp__, 0, __i__);

			omp_map_buffer_malloc(__dev_map_A__);

			omp_memcpyHostToDeviceAsync(__dev_map_A__);
			omp_print_data_map(__dev_map_A__);
			/*************************************************************************************************************************************************************/

			/***************************************************************** for B *********************************************************************/
			omp_data_map_t * __dev_map_B__ = &__data_maps__[__i__][1]; /* 1 is given by compiler here */
			omp_data_map_init_map(__dev_map_B__, &__data_map_infos__[1], __i__, __dev__, &__dev_stream__[__i__]);
			omp_map_buffer_malloc(__dev_map_B__); /* column major, marshalling needed */

			omp_memcpyHostToDeviceAsync(__dev_map_B__);
			omp_print_data_map(__dev_map_B__);

			/***************************************************************** for C *********************************************************************/
			omp_data_map_t * __dev_map_C__ = &__data_maps__[__i__][2]; /* 1 is given by compiler here */
			omp_data_map_init_map(__dev_map_C__, &__data_map_infos__[2], __i__, __dev__, &__dev_stream__[__i__]);
			omp_data_map_do_even_map(__dev_map_C__, 0, __topp__, 0, __i__);
			omp_map_buffer_malloc(__dev_map_C__);
			omp_print_data_map(__dev_map_C__);

			/***************************************************************************************************************************************************************/
			/*************************************************************************************************************************************************************/
			/* Launch CUDA kernel ... */
			long start_i, length_i;
			omp_loop_map_range(__dev_map_C__, 0, -1, -1, &start_i, &length_i);
			/* the argu for this function should be the original pointer (x in this example) and the runtime should search and retrieve the
			 * device map object
			*/
			int _threads_per_block_ = xomp_get_maxThreadsPerBlock();
			int _num_blocks_ = xomp_get_max1DBlock(length_i);
		//	printf("device: %d, range: %d:%d\n", __i__, start_i, length_i);

			OUT__2__7117_mdev__<<<_num_blocks_,_threads_per_block_, 0, __dev_stream__[__i__].systream.cudaStream>>>(length_i, n, n, (REAL *)__dev_map_A__->map_dev_ptr, (REAL *)__dev_map_B__->map_dev_ptr, (REAL *)__dev_map_C__->map_dev_ptr);

			omp_memcpyDeviceToHostAsync(__dev_map_C__);
	    }

	    omp_sync_cleanup(__num_target_devices__, __num_mapped_variables__, __dev_stream__, &__data_maps__[0][0]);
}

#if 0
/* multiple device */
/* B, C column-major partition */
void ompacc_matmul_mdev_v2(REAL *A, REAL *B, REAL *C, int n)
{
    int i, j, k;
#pragma omp target device(*) map(from:C{0:n}[0:n]>>(*)), map(to:n,A[0:n][0:n],B{0:n}[0:n]>>(*)
    for (i = 0; i < n; i++)
#pragma omp parallel for private(i,j,k) dist_iteration match_range C{}[]
        for (k = 0; k < n; k++) {
            REAL c = 0.0;
            for (j = 0; j < n; j++)
                c += A[i * n + j] * B[j * n + k];
            C[i * n + k] = c;
        }
}
#endif

void matmul_ompacc_mdev_v2(REAL *A, REAL *B, REAL *C,  int n)
{
	   /* get number of target devices specified by the programmers */
	    int __num_target_devices__ = omp_get_num_active_devices(); /*XXX: = runtime or compiler generated code */

	    printf("use %d target devices\n", __num_target_devices__);

		omp_device_t *__target_devices__[__num_target_devices__];
		/**TODO: compiler generated code or runtime call to init the __target_devices__ array */
		int __i__;
		for (__i__ = 0; __i__ < __num_target_devices__; __i__++) {
			__target_devices__[__i__] = &omp_devices[__i__]; /* currently this is simple a copy of the pointer */
		}
		/**TODO: compiler generated code or runtime call to init the topology */
		int __top_ndims__ = 1;
		int __top_dims__[__top_ndims__];
		omp_factor(__num_target_devices__, __top_dims__, __top_ndims__);
		int __top_periodic__[__top_ndims__]; __top_periodic__[0] = 0;
		omp_grid_topology_t __topology__={__num_target_devices__, __top_ndims__, __top_dims__, __top_periodic__};
		omp_grid_topology_t *__topp__ = &__topology__;

		int __num_mapped_variables__ = 3; /* XXX: need compiler output */

		omp_stream_t __dev_stream__[__num_target_devices__]; /* need to change later one for omp_stream_t struct */
		omp_data_map_info_t __data_map_infos__[__num_mapped_variables__];

		omp_data_map_info_t * __info__ = &__data_map_infos__[0];
		omp_data_map_init_info(__info__, __topp__, A, sizeof(REAL), OMP_MAP_TO, n, n, 1);
		__info__->maps = (omp_data_map_t **)alloca(sizeof(omp_data_map_t *) * __num_target_devices__);

		__info__ = &__data_map_infos__[1];
		omp_data_map_init_info(__info__, __topp__, B, sizeof(REAL), OMP_MAP_TO, n, n, 1);
		__info__->maps = (omp_data_map_t **)alloca(sizeof(omp_data_map_t *) * __num_target_devices__);

		__info__ = &__data_map_infos__[2];
		omp_data_map_init_info(__info__, __topp__, C, sizeof(REAL), OMP_MAP_FROM, n, n, 1);
		__info__->maps = (omp_data_map_t **)alloca(sizeof(omp_data_map_t *) * __num_target_devices__);

		omp_data_map_t __data_maps__[__num_target_devices__][__num_mapped_variables__];
		for (__i__ = 0; __i__ < __num_target_devices__; __i__++) {
	    		printf("=========================================== device %d ==========================================\n", __i__);
			omp_device_t * __dev__ = __target_devices__[__i__];
			omp_set_current_device(__dev__);
			omp_init_stream(__dev__, &__dev_stream__[__i__]);

			/***************** for each mapped variable has to and tofrom, if it has region mapped to this __ndev_i__ id, we need code here *******************************/
			omp_data_map_t * __dev_map_A__ = &__data_maps__[__i__][0]; /* 0 is given by compiler here */
			omp_data_map_init_map(__dev_map_A__, &__data_map_infos__[0], __i__, __dev__, &__dev_stream__[__i__]);

			omp_map_buffer_malloc(__dev_map_A__);

			omp_memcpyHostToDeviceAsync(__dev_map_A__);
			omp_print_data_map(__dev_map_A__);
			/*************************************************************************************************************************************************************/

			/***************************************************************** for B *********************************************************************/
			omp_data_map_t * __dev_map_B__ = &__data_maps__[__i__][1]; /* 1 is given by compiler here */
			omp_data_map_init_map(__dev_map_B__, &__data_map_infos__[1], __i__, __dev__, &__dev_stream__[__i__]);
			omp_data_map_do_even_map(__dev_map_B__, 1, __topp__, 0, __i__);
			omp_map_buffer_malloc(__dev_map_B__); /* column major, marshalling needed */

			omp_memcpyHostToDeviceAsync(__dev_map_B__);
			omp_print_data_map(__dev_map_B__);

			/***************************************************************** for C *********************************************************************/
			omp_data_map_t * __dev_map_C__ = &__data_maps__[__i__][2]; /* 1 is given by compiler here */
			omp_data_map_init_map(__dev_map_C__, &__data_map_infos__[2], __i__, __dev__, &__dev_stream__[__i__]);
			omp_data_map_do_even_map(__dev_map_C__, 1, __topp__, 0, __i__);
			omp_map_buffer_malloc(__dev_map_C__);
			omp_print_data_map(__dev_map_C__);

			/***************************************************************************************************************************************************************/
			/*************************************************************************************************************************************************************/
			/* Launch CUDA kernel ... */
			long start_k, length_k;
			omp_loop_map_range(__dev_map_C__, 1, -1, -1, &start_k, &length_k);
			/* the argu for this function should be the original pointer (x in this example) and the runtime should search and retrieve the
			 * device map object
			*/
			int _threads_per_block_ = xomp_get_maxThreadsPerBlock();
			int _num_blocks_ = xomp_get_max1DBlock(length_k);
	//		printf("device: %d, range: %d:%d\n", __i__, start_k, length_k);

			OUT__2__7117_mdev__<<<_num_blocks_,_threads_per_block_, 0, __dev_stream__[__i__].systream.cudaStream>>>(n, n, length_k, (REAL *)__dev_map_A__->map_dev_ptr, (REAL *)__dev_map_B__->map_dev_ptr, (REAL *)__dev_map_C__->map_dev_ptr);
			omp_memcpyDeviceToHostAsync(__dev_map_C__);
	    }

	    omp_sync_cleanup(__num_target_devices__, __num_mapped_variables__, __dev_stream__, &__data_maps__[0][0]);

}

#if 0
/* multiple device */
/* A,B, C row-column partition */
void ompacc_matmul_mdev_v3(REAL *A, REAL *B, REAL *C, int n)
{
    int i, j, k;
#pragma omp target device(*)=>(:)(:) map(from:C[0:n][0:n]>>(:)(:)), map(to:n,A[0:n]{0:n}>>(:){:},B{0:n}[0:n]>>{:}())
#pragma omp parallel for private(i,j,k) dist_iteration match_range C[]{}
    for (i = 0; i < n; i++)
#pragma omp parallel for private(i,j,k) dist_iteration match_range C{}[]
        for (k = 0; k < n; k++) {
            REAL c = 0.0;
            for (j = 0; j < n; j++)
                c += A[i * n + j] * B[j * n + k];
            C[i * n + k] = c;
        }
}
#endif

// Cannon's Matrix multiplication performs 2-D partitioned matrix-multiply.
// The implementation requires skewing.
void matmul_ompacc_mdev_v3(REAL *A, REAL *B, REAL *C,  int n)
{
	   /* get number of target devices specified by the programmers */
	    int __num_target_devices__ = omp_get_num_active_devices(); /*XXX: = runtime or compiler generated code */

	    printf("use %d target devices\n", __num_target_devices__);

		omp_device_t *__target_devices__[__num_target_devices__];
		/**TODO: compiler generated code or runtime call to init the __target_devices__ array */
		int __i__;
		for (__i__ = 0; __i__ < __num_target_devices__; __i__++) {
			__target_devices__[__i__] = &omp_devices[__i__]; /* currently this is simple a copy of the pointer */
		}
		/**TODO: compiler generated code or runtime call to init the topology */
		int __top_ndims__ = 2;
		int __top_dims__[__top_ndims__];
		omp_factor(__num_target_devices__, __top_dims__, __top_ndims__);
		int __top_periodic__[__top_ndims__]; __top_periodic__[0] = 0; __top_periodic__[1] = 0;
		omp_grid_topology_t __topology__={__num_target_devices__, __top_ndims__, __top_dims__, __top_periodic__};
		omp_grid_topology_t *__topp__ = &__topology__;

		int __num_mapped_variables__ = 3; /* XXX: need compiler output */

		omp_stream_t __dev_stream__[__num_target_devices__]; /* need to change later one for omp_stream_t struct */
		omp_data_map_info_t __data_map_infos__[__num_mapped_variables__];

		omp_data_map_info_t * __info__ = &__data_map_infos__[0];
		omp_data_map_init_info(__info__, __topp__, A, sizeof(REAL), OMP_MAP_TO, n, n, 1);
		__info__->maps = (omp_data_map_t **)alloca(sizeof(omp_data_map_t *) * __num_target_devices__);

		__info__ = &__data_map_infos__[1];
		omp_data_map_init_info(__info__, __topp__, B, sizeof(REAL), OMP_MAP_TO, n, n, 1);
		__info__->maps = (omp_data_map_t **)alloca(sizeof(omp_data_map_t *) * __num_target_devices__);

		__info__ = &__data_map_infos__[2];
		omp_data_map_init_info(__info__, __topp__, C, sizeof(REAL), OMP_MAP_FROM, n, n, 1);
		__info__->maps = (omp_data_map_t **)alloca(sizeof(omp_data_map_t *) * __num_target_devices__);

		omp_data_map_t __data_maps__[__num_target_devices__][__num_mapped_variables__];
		for (__i__ = 0; __i__ < __num_target_devices__; __i__++) {
	    		printf("=========================================== device %d ==========================================\n", __i__);
			omp_device_t * __dev__ = __target_devices__[__i__];
			omp_set_current_device(__dev__);
			omp_init_stream(__dev__, &__dev_stream__[__i__]);

			/***************** for each mapped variable has to and tofrom, if it has region mapped to this __ndev_i__ id, we need code here *******************************/
			omp_data_map_t * __dev_map_A__ = &__data_maps__[__i__][0]; /* 0 is given by compiler here */
			omp_data_map_init_map(__dev_map_A__, &__data_map_infos__[0], __i__, __dev__, &__dev_stream__[__i__]);
			omp_data_map_do_even_map(__dev_map_A__, 0, __topp__, 0, __i__);

			omp_map_buffer_malloc(__dev_map_A__);

			omp_memcpyHostToDeviceAsync(__dev_map_A__);
			omp_print_data_map(__dev_map_A__);
			/*************************************************************************************************************************************************************/

			/***************************************************************** for B *********************************************************************/
			omp_data_map_t * __dev_map_B__ = &__data_maps__[__i__][1]; /* 1 is given by compiler here */
			omp_data_map_init_map(__dev_map_B__, &__data_map_infos__[1], __i__, __dev__, &__dev_stream__[__i__]);
			omp_data_map_do_even_map(__dev_map_B__, 1, __topp__, 1, __i__);
			omp_map_buffer_malloc(__dev_map_B__); /* column major, marshalling needed */

			omp_memcpyHostToDeviceAsync(__dev_map_B__);
			omp_print_data_map(__dev_map_B__);

			/***************************************************************** for C *********************************************************************/
			omp_data_map_t * __dev_map_C__ = &__data_maps__[__i__][2]; /* 1 is given by compiler here */
			omp_data_map_init_map(__dev_map_C__, &__data_map_infos__[2], __i__, __dev__, &__dev_stream__[__i__]);
			omp_data_map_do_even_map(__dev_map_C__, 0, __topp__, 0, __i__);
			omp_data_map_do_even_map(__dev_map_C__, 1, __topp__, 1, __i__);

			omp_map_buffer_malloc(__dev_map_C__);
			omp_print_data_map(__dev_map_C__);

			/***************************************************************************************************************************************************************/
			/*************************************************************************************************************************************************************/
			/* Launch CUDA kernel ... */
			long start_i, length_i;
			long start_k, length_k;
			omp_loop_map_range(__dev_map_C__, 0, -1, -1, &start_i, &length_i);
			omp_loop_map_range(__dev_map_C__, 1, -1, -1, &start_k, &length_k);
			/* the argu for this function should be the original pointer (x in this example) and the runtime should search and retrieve the
			 * device map object
			*/
			int _threads_per_block_ = xomp_get_maxThreadsPerBlock();
			int _num_blocks_ = xomp_get_max1DBlock(length_i);
	//		printf("device: %d, C region: %d X %d\n", __i__, length_i, length_k);

			OUT__2__7117_mdev__<<<_num_blocks_,_threads_per_block_, 0, __dev_stream__[__i__].systream.cudaStream>>>(length_i, n, length_k, (REAL *)__dev_map_A__->map_dev_ptr, (REAL *)__dev_map_B__->map_dev_ptr, (REAL *)__dev_map_C__->map_dev_ptr);

			omp_memcpyDeviceToHostAsync(__dev_map_C__);
	    }

	    omp_sync_cleanup(__num_target_devices__, __num_mapped_variables__, __dev_stream__, &__data_maps__[0][0]);
}
