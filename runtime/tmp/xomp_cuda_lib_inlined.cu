#include "hip/hip_runtime.h"
/* 
CUDA and/or nvcc does not have linker for device code.
We have to put some common device functions into this file.
So the generated CUDA file can include the functions as inlined functions.

Liao 2/11/2013
*/

/*  reduction minus is handled the same way as reduction plus since we just replace the reduction variable with its local copy for each thread
    The associated statement is intact except for the variable replacement : e.g. a-=5 becomes local_a -= 5;
    in the end of each thread accumulates thread local negative values.
    At the block level, we just simply add them all to be the block level negative values
*/
/* we have to encode the type into function name since C function signature does not include parameter list! */
#define XOMP_INNER_BLOCK_REDUCTION_DEF(dtype) \
__device__ void xomp_inner_block_reduction_##dtype(dtype local_value, dtype * grid_level_results, int reduction_op) \
{ \
  /* __shared__ float* sdata[gridDim.x]; not compilable */ \
  /* block size of data, size is specified by the kernel launch parameter (3rd one) */ \
  /* shared data has to have different names for different types. Cannot reuse name across types. */ \
  extern __shared__ dtype sdata_##dtype[];  \
  sdata_##dtype[threadIdx.x] = local_value;  \
  __syncthreads(); \
  /* blockDim.x is the block size */ \
  int isEvenSize = (blockDim.x % 2 ==0); \
  /* contiguous range pattern: half folding and add */ \
  for(int offset = blockDim.x / 2; \
      offset > 0;    /* folding and add */ \
      offset >>= 1) /* offset shrinks half each time */ \
  { \
    if(threadIdx.x < offset)  \
    { \
      /* add a partial sum upstream to our own */ \
      switch (reduction_op){ \
        case XOMP_REDUCTION_PLUS: \
        case XOMP_REDUCTION_MINUS: \
            sdata_##dtype[threadIdx.x] += sdata_##dtype[threadIdx.x + offset]; \
            break; \
         /*  TODO add support for more operations*/ \
         default:  \
            { \
              /* TODO: add assertion or set hipError_t with an error code */ \
              /* cannot call a host function */ \
              /* fprintf (stderr, "Error. xomp_inner_block_reduction() unhandled reduction operation:%d\n",reduction_op); */ \
              /* assert (false); */ \
             } \
      } /* end switch */ \
    } \
    /* remember to handle the left element */ \
    if ((threadIdx.x == 0) && !isEvenSize) \
    { \
      switch (reduction_op){ \
        case XOMP_REDUCTION_PLUS: \
        case XOMP_REDUCTION_MINUS: \
          sdata_##dtype[0]+= sdata_##dtype[2*offset];  \
          break; \
        /* TODO add more operation support */  \
        default: \
          {  \
            /* TODO: add assertion or set hipError_t with an error code */  \
            /* cannot call a host function */ \
            /* fprintf (stderr, "Error. xomp_inner_block_reduction() unhandled reduction operation:%d\n",reduction_op); */ \
            /* assert (false); */ \
          } \
      } /* end switch */ \
    } \
    isEvenSize = ( offset % 2 ==0); /* prepare next round*/ \
    /* MUST wait until all threads in the block have updated their partial sums */ \
    __syncthreads(); /* sync after each folding */ \
  } \
  /* thread 0 writes the final result to the partial sum of this thread block */ \
  if(threadIdx.x == 0) \
  { \
    grid_level_results[blockIdx.x] = sdata_##dtype[0]; \
  } \
}

XOMP_INNER_BLOCK_REDUCTION_DEF(int)
XOMP_INNER_BLOCK_REDUCTION_DEF(float)
XOMP_INNER_BLOCK_REDUCTION_DEF(double)

#undef XOMP_INNER_BLOCK_REDUCTION_DEF 

/* Loop scheduling helper functions on GPUs */
//Default loop scheduling, worksharing without any schedule clause
// input upper bound is inclusive (loop normalized with <= or >=)
// output n_upper is also inclusive 
// stride is positive for incremental, negative for decremental iteration space
__device__ void XOMP_cuda_loop_default_internal(int lower, int upper, int stride, int _p_num_threads, int _p_thread_id, long* n_lower, long* n_upper)
{
  int _p_lower;
  int _p_upper;
  int _p_chunk_size;
  int addOne ; // adjustment to input and output upper bounds, depending on if they are inclusive or non-inclusive

  int isDecremental= 0;
  if (lower>upper)
    isDecremental = 1;

  // calculate loop iteration count from lower, upper and stride , 
  // adjust inclusive stride here TODO
  // no -1/+1? if upper is already an inclusive bound
  int _p_iter_count = 0;
  if (isDecremental == 1)
  {
    addOne = 1;
    //stride = 0 - stride;  // n
    if (stride >0)
    {
/*  TODO: assertion and error code  
      printf("Error: in XOMP_loop_default() of xomp.c: stride must be negative for decremental iteration. stride = %d \n ", stride);
      assert (0);
*/
    }
  }
  else // incremental
  {
    addOne = -1; // real bound should be non-inclusive - 1
    if (stride <0)
    {
/* TODO assertion and error code
      printf("Error: in XOMP_loop_default() of xomp.c: stride must be positive for incremental iteration. stride = %d \n ", stride);
      assert (0);
*/
    }
  }
  // addOne is needed here only if the input upper bound is non-inclusive
  // we use loop normalization to ensure upper bounds are inclusive already.
  // So we don't need addOne here anymore
  //_p_iter_count = ( stride + addOne + upper -  lower) /  stride;
  _p_iter_count = ( stride + upper -  lower) /  stride;

  // calculate a proper chunk size
  // two cases: evenly divisible  20/5 =4
  //   not evenly divisible 20/3= 6
  // Initial candidate  
 // get number of threads of this thread block, 1-D case
//  int _p_num_threads = blockDim.x ;  
  //int _p_num_threads = omp_get_num_threads();

  _p_chunk_size = _p_iter_count / _p_num_threads;
  int _p_ck_temp = _p_chunk_size * _p_num_threads != _p_iter_count;

  _p_chunk_size = _p_ck_temp + _p_chunk_size;

  // decide on the lower and upper bound for the current thread
  //int _p_thread_id = threadIdx.x ;
  //int _p_thread_id = omp_get_thread_num();
//  printf("inside xomp_loop_default(): _p_thread_id =%d\n", _p_thread_id);
  _p_lower =  lower + _p_chunk_size * _p_thread_id *  stride;

  //addOne is needed here if the output upper bound is inclusive
  // -1 if the output n_upper is an inclusive bound, 
  // we do use inclusive in the final normalized loop
  _p_upper = _p_lower + _p_chunk_size *  stride + addOne;

  // adjust inclusive stride here 
  // addOne is needed if the input upper bound is non-inclusive
  // no -1/+1 since upper is already an inclusive bound
  if (isDecremental == 1)
    _p_upper = (_p_upper > (upper ) ?_p_upper : (upper ));
    //_p_upper = (_p_upper > (upper + addOne) ?_p_upper : (upper + addOne));
  else
    _p_upper = (_p_upper < upper ?_p_upper : upper);
    //_p_upper = (_p_upper < (upper + addOne) ?_p_upper : (upper + addOne));

  *n_lower = _p_lower;
  *n_upper = _p_upper;
//  printf("inside xomp_loop_default(): _p_lower=%d, _p_upper=%d\n", _p_lower,_p_upper);
}

/* This is the top level function which decides on the lower and upper bounds for the current thread id and current block id
  It internally calls XOMP_cuda_loop_default_internal twice to figure this out: 
  once for the block's bounds, the other for the thread's bounds
*/
__device__ void XOMP_accelerator_loop_default(int lower, int upper, int stride, long* n_lower, long* n_upper)
{
/* Obtain the lower and upper bounds for the current 1-D thread block*/
 /*  the iteration bounds for this entire thread block */
  long lower_for_block, upper_for_block;
  XOMP_cuda_loop_default_internal (lower, upper, stride, gridDim.x, blockIdx.x, &lower_for_block, &upper_for_block);

 /* now focus on the bounds of the current thread of the current block */
  XOMP_cuda_loop_default_internal (lower_for_block, upper_for_block, stride, blockDim.x, threadIdx.x, n_lower, n_upper);
}



