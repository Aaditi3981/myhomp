#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <omp.h>
#include <sys/time.h>
#include "homp.h"
#include "bm2d.h"
#include "xomp_cuda_lib_inlined.cu"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

/* this works only for 1-d row-wise partition */
__global__ void bm2d_nvgpu_kernel(int start_n, int len_n, long n, long m, int u_dimX, int u_dimY, REAL *u, REAL *uold, int maxwin, int coeff_dimX, REAL *coeff) {
    long ix, iy, ir;
    long ixy;
    long ixy_lower, ixy_upper;
    hiprandState_t state;

    /* we have to initialize the state */
    hiprand_init(0, /* the seed controls the sequence of random values that are produced */
                  0, /* the sequence number is only important with multiple cores */
                  0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
                  &state);

    // variables for adjusted loop info considering both original chunk size and step(strip)
    long _dev_loop_chunk_size;
    long _dev_loop_sched_index;
    long _dev_loop_stride;

    // 1-D thread block:
    long _dev_thread_num = gridDim.x * blockDim.x;
    long _dev_thread_id = blockDim.x * blockIdx.x + threadIdx.x;

    //TODO: adjust bound to be inclusive later
    long orig_start = 0;
    long orig_end = len_n * m; /* Linearized iteration space */
    long orig_step = 1;
    long orig_chunk_size = 1;

    XOMP_static_sched_init(orig_start, orig_end, orig_step, orig_chunk_size, _dev_thread_num, _dev_thread_id, \
      &_dev_loop_chunk_size, &_dev_loop_sched_index, &_dev_loop_stride);

    //XOMP_accelerator_loop_default (1, (n-1)*(m-1)-1, 1, &_dev_lower, &_dev_upper);
    while (XOMP_static_sched_next(&_dev_loop_sched_index, orig_end, orig_step, _dev_loop_stride, _dev_loop_chunk_size,
                                  _dev_thread_num, _dev_thread_id, &ixy_lower, &ixy_upper)) {
        for (ixy = ixy_lower; ixy <= ixy_upper; ixy++) {
            ix = ixy / m;
            iy = ixy % m;
            if (!(ix>=start_n && ix<=start_n+len_n-1 && iy>=0 && iy<=m-1)) continue;
            int radius = hiprand_uniform_double(&state) * maxwin;
            if (radius < 1) continue;

            int count = 4*radius+1;
#ifdef SQUARE_SETNCIL
	        count = coeff_dimX * coeff_dimX;
#endif
            long offset = (ix+radius)*u_dimY+radius+iy;
            REAL *temp_u = &u[offset];
            REAL *temp_uold = &uold[offset];
            REAL result = temp_uold[0] * coeff[0];
            /* 2/4 way loop unrolling */
            for (ir = 1; ir <= radius; ir++) {
                result += coeff[ir] * temp_uold[ir];                //horizontal right
                result += coeff[-ir] * temp_uold[-ir];                  // horizontal left
                result += coeff[-ir * coeff_dimX] * temp_uold[-ir * u_dimY]; //vertical up
                result += coeff[ir * coeff_dimX] * temp_uold[ir * u_dimY]; // vertical bottom
#ifdef SQUARE_SETNCIL
				result += coeff[-ir*coeff_dimX-ir] * temp_uold[-ir * u_dimY-ir] // left upper corner
				result += coeff[-ir*coeff_dimX+ir] * temp_uold[-ir * u_dimY+ir] // right upper corner
				result += coeff[ir*coeff_dimX-ir] * temp_uold[ir * u_dimY]-ir] // left bottom corner
				result += coeff[ir*coeff_dimX+ir] * temp_uold[ir * u_dimY]+ir] // right bottom corner
#endif
            }
            *temp_u = result/count;
        }
    }
}

void bm2d_nvgpu_cuda_wrapper(omp_offloading_t *off, int start, int len, long n, long m, int u_dimX, int u_dimY, REAL *u, REAL *uold, int maxwin, int coeff_dimX, REAL *coeff) {
    dim3 threads_per_team(16, 16);
    dim3 teams_per_league((len + threads_per_team.x - 1) / threads_per_team.x,
                          (m + threads_per_team.y - 1) / threads_per_team.y); /* we assume dividable */
    bm2d_nvgpu_kernel<<<teams_per_league, threads_per_team, 0, off->stream->systream.cudaStream>>>
                                                                       (start, len, n, m, u_dimX, u_dimY, u, uold, maxwin, coeff_dimX, coeff);
}


