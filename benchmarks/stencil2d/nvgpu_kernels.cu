#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <omp.h>
#include <sys/time.h>
#include "homp.h"
#include "stencil2d.h"

#include "xomp_cuda_lib_inlined.cu"

//#define LOOP_COLLAPSE 1
#if !LOOP_COLLAPSE
__global__ void stencil2d_nvgpu_kernel(int start_n, int len_n, long n, long m, int u_dimX, int u_dimY, REAL *u, REAL *uold, int radius, int coeff_dimX, REAL *coeff) {
    long ix_lower, ix_upper;
    XOMP_accelerator_loop_default(start_n, len_n, 1, &ix_lower, &ix_upper);
    long ix, iy, ir;
    for (ix = ix_lower; iy <= ix_upper; ix++) {
        for (iy = 0; iy < m; iy++) {
            REAL *temp_u = &u[(ix + radius) * u_dimY + radius+iy];
            REAL *temp_uold = &uold[(ix + radius) * u_dimY + radius+iy];
            REAL result = temp_uold[0] * coeff[0];
            /* 2/4 way loop unrolling */
            for (ir = 1; ir <= radius; ir++) {
                result += coeff[ir] * temp_uold[ir];                //horizontal right
                result += coeff[-ir] * temp_uold[-ir];                  // horizontal left
                result += coeff[-ir * coeff_dimX] * temp_uold[-ir * u_dimY]; //vertical up
                result += coeff[ir * coeff_dimX] * temp_uold[ir * u_dimY]; // vertical bottom
#ifdef SQUARE_SETNCIL
				result += coeff[-ir*coeff_dimX-ir] * temp_uold[-ir * u_dimY-ir] // left upper corner
				result += coeff[-ir*coeff_dimX+ir] * temp_uold[-ir * u_dimY+ir] // right upper corner
				result += coeff[ir*coeff_dimX-ir] * temp_uold[ir * u_dimY]-ir] // left bottom corner
				result += coeff[ir*coeff_dimX+ir] * temp_uold[ir * u_dimY]+ir] // right bottom corner
#endif
            }
            *temp_u = result;
        }
    }
}
#else

__global__ void stencil2d_nvgpu_kernel(int start_n, int len_n, long n, long m, int u_dimX, int u_dimY, REAL *u, REAL *uold, int radius, REAL *coeff) {
    long ix_lower, ix_upper;
    XOMP_accelerator_loop_default(start_n, len_n, 1, &ix_lower, &ix_upper);
    long ix;
    for (ix = ix_lower; iy <= ix_upper; ix++) {
        for (iy = 0; iy < m; iy++) {
            REAL *temp_u = &u[(ix + radius) * u_dimY + radius+iy];
            REAL *temp_uold = &uold[(ix + radius) * u_dimY + radius+iy];
            REAL result = temp_uold[0] * coeff[0];
            /* 2/4 way loop unrolling */
            for (ir = 1; ir <= radius; ir++) {
                result += coeff[ir] * temp_uold[ir];                //horizontal right
                result += coeff[-ir] * temp_uold[-ir];                  // horizontal left
                result += coeff[-ir * coeff_dimX] * temp_uold[-ir * u_dimY]; //vertical up
                result += coeff[ir * coeff_dimX] * temp_uold[ir * u_dimY]; // vertical bottom
#ifdef SQUARE_SETNCIL
				result += coeff[-ir*coeff_dimX-ir] * temp_uold[-ir * u_dimY-ir] // left upper corner
				result += coeff[-ir*coeff_dimX+ir] * temp_uold[-ir * u_dimY+ir] // right upper corner
				result += coeff[ir*coeff_dimX-ir] * temp_uold[ir * u_dimY]-ir] // left bottom corner
				result += coeff[ir*coeff_dimX+ir] * temp_uold[ir * u_dimY]+ir] // right bottom corner
#endif
            }
            *temp_u = result;
        }
    }
}




__global__ void OUT__1__10550__(long n, long m, REAL omega, REAL ax, REAL ay, REAL b, REAL *_dev_u, REAL *_dev_f,
                                REAL *_dev_uold,
                                long uold_m, int uold_0_offset, int uold_1_offset, int start_i, int start_j,
                                REAL *_dev_per_block_error) {
    long _dev_i;
    long ij;
    long _p_j;
    long _dev_lower, _dev_upper;

    REAL _p_error;
    _p_error = 0;
    REAL _p_resid;

    // variables for adjusted loop info considering both original chunk size and step(strip)
    long _dev_loop_chunk_size;
    long _dev_loop_sched_index;
    long _dev_loop_stride;

    // 1-D thread block:
    long _dev_thread_num = gridDim.x * blockDim.x;
    long _dev_thread_id = blockDim.x * blockIdx.x + threadIdx.x;

    //TODO: adjust bound to be inclusive later
    long orig_start = start_i * m;
    long orig_end = (n - start_i) * m - 1; /* Linearized iteration space */
    long orig_step = 1;
    long orig_chunk_size = 1;

    XOMP_static_sched_init(orig_start, orig_end, orig_step, orig_chunk_size, _dev_thread_num, _dev_thread_id, \
      &_dev_loop_chunk_size, &_dev_loop_sched_index, &_dev_loop_stride);

    //XOMP_accelerator_loop_default (1, (n-1)*(m-1)-1, 1, &_dev_lower, &_dev_upper);
    while (XOMP_static_sched_next(&_dev_loop_sched_index, orig_end, orig_step, _dev_loop_stride, _dev_loop_chunk_size,
                                  _dev_thread_num, _dev_thread_id, &_dev_lower, &_dev_upper)) {
        for (ij = _dev_lower; ij <= _dev_upper; ij++) {
            _dev_i = ij / (m - 1);
            _p_j = ij % (m - 1);

            if (_dev_i >= start_i && _dev_i < (n) && _p_j >= 1 &&
                _p_j < (m - 1)) // must preserve the original boudary conditions here!!
            {
                _p_resid = (((((ax * (_dev_uold[(_dev_i - 1 + uold_0_offset) * uold_m + _p_j + uold_1_offset] +
                                      _dev_uold[(_dev_i + 1 + uold_0_offset) * uold_m + _p_j + uold_1_offset])) +
                               (ay * (_dev_uold[(_dev_i + uold_0_offset) * uold_m + (_p_j - 1 + uold_1_offset)] +
                                      _dev_uold[(_dev_i + uold_0_offset) * uold_m + (_p_j + 1 + uold_1_offset)]))) +
                              (b * _dev_uold[(_dev_i + uold_0_offset) * uold_m + _p_j + uold_1_offset])) -
                             _dev_f[(_dev_i + uold_0_offset) * uold_m + _p_j + uold_1_offset]) / b);
                _dev_u[_dev_i * uold_m + _p_j] = (_dev_uold[(_dev_i + uold_0_offset) * uold_m + _p_j + uold_1_offset] -
                                                  (omega * _p_resid));
                _p_error = (_p_error + (_p_resid * _p_resid));
            }
        }
    }

    xomp_inner_block_reduction_float(_p_error, _dev_per_block_error, 6);
}

#endif /* LOOP_CLAPSE */