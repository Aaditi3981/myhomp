#include "hip/hip_runtime.h"
#include "matvec.h"
#include <homp.h>
#include "hipblas.h"

#include "xomp_cuda_lib_inlined.cu"
__global__ void matverc_nvgpu_cuda_kernel(long n, long start_n, long length_n,REAL *_dev_a,REAL *_dev_x,REAL *_dev_y)
{
  int i,j;
  long _dev_lower;
  long  _dev_upper;
  long _dev_loop_chunk_size;
  long _dev_loop_sched_index;
  long _dev_loop_stride;
  int _dev_thread_num = getCUDABlockThreadCount(1);
  int _dev_thread_id = getLoopIndexFromCUDAVariables(1);
  XOMP_static_sched_init(start_n,start_n + length_n - 1,1,1,_dev_thread_num,_dev_thread_id,&_dev_loop_chunk_size,&_dev_loop_sched_index,&_dev_loop_stride);
  while(XOMP_static_sched_next(&_dev_loop_sched_index,start_n + length_n - 1,1,_dev_loop_stride,_dev_loop_chunk_size,_dev_thread_num,_dev_thread_id,&_dev_lower,&_dev_upper))
    for (i = _dev_lower; i <= _dev_upper; i += 1) {
        for (j = 0; j<n; j++)
         _dev_y[i] += _dev_a[i*n+j] * _dev_x[j];
//		printf("x[%d]: %f, y[%d]: %f\n", i, x[i], i, y[i]);
    }
}

void matvec_nvgpu_cuda_wrapper(omp_offloading_t *off, long n, long start_n, long length_n,REAL *a,REAL *x,REAL *y)
{
int threads_per_team = omp_get_optimal_threads_per_team(off->dev);
		int teams_per_league = omp_get_optimal_teams_per_league(off->dev, threads_per_team, length_n);
       // matverc_nvgpu_cuda_kernel<<<teams_per_league,threads_per_team, 0, off->stream->systream.cudaStream>>>(n, start_n, length_n,(REAL *)a,(REAL *)x,(REAL *)y);
        //hipblasHandle_t handle;
        //hipblasCreate(&handle);
        const float alpha = 1.0f;
        const float beta  = 0.0f;
        hipblasSgemv((hipblasHandle_t)off->dev->cublas_handle, HIPBLAS_OP_N, length_n - start_n, n,  &alpha, a, n, x, 1, &beta, y, 1);
        //hipblasDestroy(handle);
}
